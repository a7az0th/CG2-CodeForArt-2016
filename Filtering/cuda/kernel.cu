#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "utils.cuh"

__device__ float4 convolve(int i, int j, const float *kernel, int k, const uint32* img, int imgWidth, int imgHeight) {
	float4 res = make_float4(0.f, 0.f, 0.f, 1.f);

	for (int u = -k; u <= k; u++) {
		for (int v = -k; v <= k; v++) {
			const int ix = clamp(i - u, 0, imgWidth -1);
			const int iy = clamp(j - v, 0, imgHeight - 1);
			const float4 col = toFloat4(img[imgWidth * iy + ix]);
			res +=  col * kernel[(u+k)*(2*k+1) + (v+k)];
		}
	}
	return res;
}

__global__ void convolutionKernel(uint32 *glBuff, const uint32* srcBuff, int imgWidth, int imgHeight, const float *convKernel, int nbhd) {

	// global thread indices for x and y
	const int ix = blockIdx.x * blockDim.x + threadIdx.x;
	const int iy = blockIdx.y * blockDim.y + threadIdx.y;

	// global mem address of this thread
	const int globalID = iy * imgWidth + ix;

	if (nbhd > MAX_KERNEL_RADIUS) {
		if (ix < imgWidth && iy < imgHeight) {
			float4 col = convolve(ix, iy, convKernel, nbhd, srcBuff, imgWidth, imgHeight);
			col = clamp(col, 0.f, 1.f);
			glBuff[globalID] = toInt(col);
		}
	} else {
		convolveShared(glBuff, srcBuff, imgWidth, imgHeight, convKernel, nbhd);
	}
}

extern "C"
void runCudaKernel(void* glBuffer, void* deviceBuffer, int width, int height, void *convKernel, int nbhd) {
	dim3 threads(TILE_DIM, TILE_DIM);
	dim3 grid(divUp(width, TILE_DIM), divUp(height, TILE_DIM));

	convolutionKernel<<<grid, threads>>>((uint32*)glBuffer, (uint32*)deviceBuffer, width, height, (float*)convKernel, nbhd);
}
